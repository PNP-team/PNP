#include "ATen/native/biginteger/cuda/sppark-ntt/ntt.cuh"

namespace at { 
namespace native {
    
void bit_rev(BLS12_381_Fr_G1* d_out, const BLS12_381_Fr_G1* d_inp,
                        uint32_t lg_domain_size, stream_t& stream)
{
    //assert(lg_domain_size <= MAX_LG_DOMAIN_SIZE);
    TORCH_CHECK(lg_domain_size <= MAX_LG_DOMAIN_SIZE, "NTT length cannot exceed MAX_LG_DOMAIN_SIZE!");
    size_t domain_size = (size_t)1 << lg_domain_size;

    if (domain_size <= WARP_SZ)
        bit_rev_permutation
            <<<1, domain_size, 0, stream>>>
            (d_out, d_inp, lg_domain_size);
    else if (d_out == d_inp)
        bit_rev_permutation
            <<<domain_size/WARP_SZ, WARP_SZ, 0, stream>>>
            (d_out, d_inp, lg_domain_size);
    else if (domain_size < 1024)
        bit_rev_permutation_aux
            <<<1, domain_size / 8, domain_size * sizeof(BLS12_381_Fr_G1), stream>>>
            (d_out, d_inp, lg_domain_size);
    else
        bit_rev_permutation_aux
            <<<domain_size / 1024, 1024 / 8, 1024 * sizeof(BLS12_381_Fr_G1), stream>>>
            (d_out, d_inp, lg_domain_size);

    //CUDA_OK(hipGetLastError());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void LDE_powers(BLS12_381_Fr_G1* inout, bool innt, bool bitrev,
                        uint32_t lg_domain_size, uint32_t lg_blowup,
                        stream_t& stream, bool ext_pow)
{
    size_t domain_size = (size_t)1 << lg_domain_size;
    const auto gen_powers =
        NTTParameters::all(innt)[stream]->partial_group_gen_powers;

    if (domain_size < WARP_SZ)
        LDE_distribute_powers<<<1, domain_size, 0, stream>>>
                                (inout, lg_blowup, bitrev, gen_powers, ext_pow);
    else if (domain_size < 512)
        LDE_distribute_powers<<<domain_size / WARP_SZ, WARP_SZ, 0, stream>>>
                                (inout, lg_blowup, bitrev, gen_powers, ext_pow);
    else
        LDE_distribute_powers<<<domain_size / 512, 512, 0, stream>>>
                                (inout, lg_blowup, bitrev, gen_powers, ext_pow);

    //CUDA_OK(hipGetLastError());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

void NTT_internal(BLS12_381_Fr_G1* d_inout,
                    BLS12_381_Fr_G1 (*partial_twiddles)[WINDOW_SIZE],
                    BLS12_381_Fr_G1* radix_twiddles,
                    BLS12_381_Fr_G1* radix_middles,
                    BLS12_381_Fr_G1 (*partial_group_gen_powers)[WINDOW_SIZE],
                    uint32_t* Domain_size_inverse,
                    uint32_t lg_domain_size,
                    InputOutputOrder order, Direction direction,
                    Type type, stream_t& stream,
                    bool coset_ext_pow)
{
    // Pick an NTT algorithm based on the input order and the desired output
    // order of the data. In certain cases, bit reversal can be avoided which
    // results in a considerable performance gain.

    const bool intt = direction == Direction::inverse;
    //const auto& ntt_parameters = *NTTParameters::all(intt)[stream];
    //NTTParameters ntt_parameters(intt,stream);

    //const auto& ntt_parameters = parameters.all()[stream];
    bool bitrev;
    Algorithm algorithm;

    switch (order) {
        case InputOutputOrder::NN:
            bit_rev(d_inout, d_inout, lg_domain_size, stream);
            bitrev = true;
            algorithm = Algorithm::CT;
            break;
        case InputOutputOrder::NR:
            bitrev = false;
            algorithm = Algorithm::GS;
            break;
        case InputOutputOrder::RN:
            bitrev = true;
            algorithm = Algorithm::CT;
            break;
        case InputOutputOrder::RR:
            bitrev = true;
            algorithm = Algorithm::GS;
            break;
        default:
            assert(false);
    }

    if (!intt && type == Type::coset)
        LDE_powers(d_inout, intt, bitrev, lg_domain_size, 0, stream,
                    coset_ext_pow);

    switch (algorithm) {
        case Algorithm::GS:
            GS_NTT(d_inout, lg_domain_size, intt, stream,
                   partial_twiddles, 
                   radix_twiddles, radix_middles, 
                   partial_group_gen_powers, Domain_size_inverse);
            break;
        case Algorithm::CT:
            CT_NTT(d_inout, lg_domain_size, intt, stream,
                   partial_twiddles, 
                   radix_twiddles, radix_middles, 
                   partial_group_gen_powers, Domain_size_inverse);
            break;
    }

    if (intt && type == Type::coset)
        LDE_powers(d_inout, intt, !bitrev, lg_domain_size, 0, stream,
                    coset_ext_pow);

    if (order == InputOutputOrder::RR)
        bit_rev(d_inout, d_inout, lg_domain_size, stream);
}

void Base(const gpu_t& gpu, BLS12_381_Fr_G1* inout,
          BLS12_381_Fr_G1 (*partial_twiddles)[WINDOW_SIZE],
          BLS12_381_Fr_G1* radix_twiddles,
          BLS12_381_Fr_G1* radix_middles,
          BLS12_381_Fr_G1 (*partial_group_gen_powers)[WINDOW_SIZE],
          uint32_t* Domain_size_inverse,
          uint32_t lg_domain_size,
          InputOutputOrder order, Direction direction, Type type, bool coset_ext_pow)
{
    TORCH_CHECK(lg_domain_size != 0, "NTT Length cannot be 0!");
    // if (lg_domain_size == 0)
    //     return RustError{hipSuccess};

    //try {

    gpu.select();
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    size_t domain_size = (size_t)1 << lg_domain_size;

    // hipEvent_t start, stop;
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    
    NTT_internal(inout, 
                 partial_twiddles, radix_twiddles, radix_middles,
                 partial_group_gen_powers, Domain_size_inverse,
                 lg_domain_size, order, direction, type, gpu,
                 coset_ext_pow);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    gpu.sync();

    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);

    // float elapsed;
    // hipEventElapsedTime(&elapsed, start, stop);

    // std::cout << "NTT_internal: " << elapsed << " ms" << std::endl;


    //} 
//  catch (const cuda_error& e) {
//         gpu.sync();
// #ifdef TAKE_RESPONSIBILITY_FOR_ERROR_MESSAGE
//         return RustError{e.code(), e.what()};
// #else
//         return RustError{e.code()};
// #endif
//     }

//     return RustError{hipSuccess};
}

void compute_ntt(size_t device_id, BLS12_381_Fr_G1* inout, 
                 BLS12_381_Fr_G1 (*partial_twiddles)[WINDOW_SIZE],
                 BLS12_381_Fr_G1* radix_twiddles,
                 BLS12_381_Fr_G1* radix_middles,
                 BLS12_381_Fr_G1 (*partial_group_gen_powers)[WINDOW_SIZE],
                 uint64_t* Domain_size_inverse,
                 uint32_t lg_domain_size,
                 InputOutputOrder ntt_order,
                 Direction ntt_direction,
                 Type ntt_type)
{
    auto& gpu = select_gpu(device_id);

    Base(gpu, inout, 
         partial_twiddles, radix_twiddles, radix_middles,
         partial_group_gen_powers, (uint32_t *)Domain_size_inverse,
         lg_domain_size,
         ntt_order, ntt_direction, ntt_type);
}

}//namespace native
}//namespace at